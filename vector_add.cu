
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

// VECTOR ADDITION ON CUDA CORES
/*
__global__ keyword:

functions that are called from host but executed in the device, MUST BE VOID

*/
__global__ void vectorAdd(int* a, int* b, int* c, int n){
}


typedef struct arr{
    int* a;
    int* b;
    int* c;
} arr;

int main(){
    int n = 1<<16; // 2^16 
    
    size_t bytes = sizeof(int)*n; // amt of memory

    arr host{    // host 
        (int*)malloc(bytes),
        (int*)malloc(bytes),
        (int*)malloc(bytes)
    };

    arr device;
    // device has its own memory 
    // allocated to vram
    // remember to do cudaFree
    hipMalloc(&device.a,bytes); // mallocs space on the device
    hipMalloc(&device.b,bytes);        
    hipMalloc(&device.c,bytes);     

    /*
    DEVICE: gpu
    HOST: cpu
    */



    hipMemcpy(device.a, host.a, bytes, hipMemcpyHostToDevice); // a, b, annd where to memcpy   
    hipMemcpy(device.b, host.b, bytes, hipMemcpyHostToDevice); // a, b, annd where to memcpy   
    hipMemcpy(device.c, host.c, bytes, hipMemcpyHostToDevice); // a, b, annd where to memcpy   




    return 0;
}