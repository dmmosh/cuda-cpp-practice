
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <assert.h>

// VECTOR ADDITION ON CUDA CORES
/*
__global__ keyword:
CUDA kernel
functions that are called from host but executed in the device, MUST BE VOID

*/
__global__ void vectorAdd(int* a, int* b, int* c, int n){
    // 1 thread PER ELEMENT that gets added

    /*
    BLOCKID:
    block number 
    start at 0

    BLOCKDIM:
    block size (constant, 256)

    THREADID:
    which thred in the thread block were in 
    starts at 0

    all are in x, y, z dimensions 
    
    */
    int tid = (blockIdx.x* blockDim.x) + threadIdx.x;

    if (tid<n){
        c[tid] = a[tid] + b[tid];
    }

}

void rand_0_99(int* arr, int n){
    for(int i=0; i<n; i++){
      arr[i] = (rand() % (99 - 1)) + 0;
    }
}

typedef struct arr{
    int* a;
    int* b;
    int* c;
} arr;

int main(){
    srand(time(NULL));
    int n = 1<<16; // 2^16 , number of elements
    
    size_t bytes = sizeof(int)*n; // amt of memory

    arr host{    // host 
        (int*)malloc(bytes),
        (int*)malloc(bytes),
        (int*)malloc(bytes)
    };

    arr device;
    // device has its own memory 
    // allocated to vram
    // remember to do cudaFree
    hipMalloc(&device.a,bytes); // mallocs space on the device
    hipMalloc(&device.b,bytes);        
    hipMalloc(&device.c,bytes);     

    /*
    DEVICE: gpu
    HOST: cpu
    */
    rand_0_99(host.a, n);
    rand_0_99(host.b, n);


    // memcpy
    // cudaMemcpyaTob (a to b )
    hipMemcpy(device.a, host.a, bytes, hipMemcpyHostToDevice); // a, b, annd where to memcpy   
    hipMemcpy(device.b, host.b, bytes, hipMemcpyHostToDevice); // a, b, annd where to memcpy   
    hipMemcpy(device.c, host.c, bytes, hipMemcpyHostToDevice); // a, b, annd where to memcpy   

    int NUM_THREADS = 256; // amt of threads, multiple of 32
    int NUM_WARPS = NUM_THREADS/32;  // 8
    int NUM_BLOCKS  = n / NUM_THREADS;
    vectorAdd<<<NUM_BLOCKS,NUM_THREADS>>>(device.a,device.b,device.c,n);

    hipMemcpy(host.c, device.c,bytes, hipMemcpyDeviceToHost); // device: gpu, host: cpu


    for(int i = 0; i<n; i++){
        printf("%i ", host.c[i]);
    }

    hipFree(device.a);
    hipFree(device.b);
    hipFree(device.c);


    // VALGRIND TO CHECK MEM
    free(host.a);
    free(host.b);
    free(host.c);

    return 0;
}