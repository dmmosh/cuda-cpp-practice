
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

// VECTOR ADDITION ON CUDA CORES
/*
__global__ keyword:

functions that are called from cpu but executed in the gpu, MUST BE VOID

*/
__global__ void vectorAdd(int* a, int* b, int* c, int n){
}


typedef struct arr{
    int* a;
    int* b;
    int* c;
} arr;

int main(){
    int n = 1<<16; // 2^16 
    
    size_t bytes = sizeof(int)*n; // amt of memory

    arr cpu{    // cpu 
        (int*)malloc(bytes),
        (int*)malloc(bytes),
        (int*)malloc(bytes)
    };

    arr gpu;
    // gpu has its own memory 
    hipMalloc(&gpu.a,bytes); // mallocs space on the gpu
    hipMalloc(&gpu.b,bytes);        
    hipMalloc(&gpu.c,bytes);        

    return 0;
}