
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <memory>
#include <cudnn_adv.h>
/*
CUDA PRACTICE 

https://youtu.be/86FAWCzIe_4

typical CUDA flow:
1 cpu allocates memory
2 cpu copies data to gpu
3 cpu launched kernel on gpu 
4 cpu copies from gpu back to cpu to do something with it

cpu: host, execuses FUNCTIONS
gpu: device, execuses KERNELS (functions)
*/


/*
naming schemes:
h_A: cpu with variable name A
d_A: gpu with variable name A

*/
__global__ void make_arr(int* arr){ // kernel, runs on the GPU and can be run by cpu
    //usually void and do things on the argument
    return;
}

__global__ void free_arr(int* arr){
    return;
}

__device__ void gpu_only(void){ // functions ONLY for the gpu

}

__host__ void hello(void){ // function that runs on the CPU (dont need, implicitly runs on cpu)


}

int main(){
    std::cout << "fdsddcscd";
    int* arr = NULL;
    std::cout<<arr;
    
    return 0;
}