
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <memory>
#include <cudnn_adv.h>
/*
CUDA PRACTICE 

https://youtu.be/86FAWCzIe_4

typical CUDA flow:
1 cpu allocates memory
2 cpu copies data to gpu
3 cpu launched kernel on gpu 
4 cpu copies from gpu back to cpu to do something with it

cpu: host, execuses FUNCTIONS
gpu: device, execuses KERNELS (functions)
*/


/*
naming schemes:
h_A: cpu with variable name A
d_A: gpu with variable name A

*/

/*
hirearchy:
each kernel (gpu function) is a thread

threads -> blocks -> grids 
aka 
kernel executed as a grid of blocks of threads


*/
__global__ void print_pos(void){ // kernel, runs on the GPU and can be run by cpu
    //usually void and do things on the argument
    //std::cout << "THREAD DIMS:\n";
    //std::cout << threadIdx.x << '\t' << threadIdx.y << '\t' << threadIdx.z << '\n';
    printf("THREAD: %i\t%i\t%i\tBLOCK: %i\t%i\t%i\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z); //ONLY PRINTF WORKS NOT STD COUT 
    
    
    return;
}

__global__ void free_arr(int* arr){
    return;
}

__device__ void gpu_only(void){ // functions ONLY for the gpu

}

__host__ void hello(void){ // function that runs on the CPU (dont need, implicitly runs on cpu)


}

int main(){
    int t_x = 4, 
        t_y = 4, 
        t_z = 4;

    dim3 thread_dim(t_x, t_y, t_z);

    int b_x = 2,
        b_y = 3,
        b_z = 4;
    
    dim3 block_dim(b_x, b_y, b_z);
    
    int t_per_block = t_x*t_y*t_z; // threads per block
    int b_per_grid = b_x*b_y*b_z; // blocks per grid

    int total_threads = t_per_block*b_per_grid;

    int* arr = NULL;


    print_pos<<<block_dim, thread_dim>>>();
    hipDeviceSynchronize();
    return 0;
}